#include "hip/hip_runtime.h"
#define NO_FREETYPE

#include <cmath>
#include <chrono>
#include <iostream>
#include <pngwriter.h>
#include <cstdlib>

#ifdef NO_FREETYPE
using namespace std;
#endif

const float colors[41][3] = {
        {1.0,  1.0,   1.0},
        {1.0,  1.0,   1.0},
        {1.0,  1.0,   1.0},
        {1.0,  1.0,   1.0},
        {1.0,  1.0,   1.0},
        {1.0,  0.7,   1.0},
        {1.0,  0.7,   1.0},
        {1.0,  0.7,   1.0},
        {0.97, 0.5,   0.94},
        {0.97, 0.5,   0.94},
        {0.94, 0.25,  0.88},
        {0.94, 0.25,  0.88},
        {0.91, 0.12,  0.81},
        {0.88, 0.06,  0.75},
        {0.85, 0.03,  0.69},
        {0.82, 0.015, 0.63},
        {0.78, 0.008, 0.56},
        {0.75, 0.004, 0.50},
        {0.72, 0.0,   0.44},
        {0.69, 0.0,   0.37},
        {0.66, 0.0,   0.31},
        {0.63, 0.0,   0.25},
        {0.60, 0.0,   0.19},
        {0.56, 0.0,   0.13},
        {0.53, 0.0,   0.06},
        {0.5,  0.0,   0.0},
        {0.47, 0.06,  0.0},
        {0.44, 0.12,  0},
        {0.41, 0.18,  0.0},
        {0.38, 0.25,  0.0},
        {0.35, 0.31,  0.0},
        {0.31, 0.38,  0.0},
        {0.28, 0.44,  0.0},
        {0.25, 0.50,  0.0},
        {0.22, 0.56,  0.0},
        {0.19, 0.63,  0.0},
        {0.16, 0.69,  0.0},
        {0.13, 0.75,  0.0},
        {0.06, 0.88,  0.0},
        {0.03, 0.94,  0.0},
        {0.0,  0.0,   0.0}
};

__global__ void cudaMandelbrot(float x0, float y0, float x1, float y1, int width, int height, int iterationsCount, int *data) {
    float dX = (x1 - x0) / float(width - 1);
    float dY = (y1 - y0) / float(height - 1);
    int i;
    float x, y, Zx, Zy, tZx, tZy;
    int idX = int(threadIdx.x + blockIdx.x * blockDim.x);
    float tmpWidth, tmpHeight;
    int size = height * width;

    if (idX < size) {
        tmpWidth = (float) idX / (float) size;
        tmpHeight = float(idX % size);
        x = x0 + dX * tmpHeight;
        y = y0 + dY * tmpWidth;
        Zx = x;
        Zy = y;
        i = 0;

        while (i < iterationsCount && ((Zx * Zx + Zy * Zy) < 4)) {
            tZx = Zx * Zx - Zy * Zy + x;
            tZy = 2 * Zx * Zy + y;
            Zx = tZx;
            Zy = tZy;
            i++;
        }

        data[idX] = i;
    }
}

__global__ void cudaMandelbrot2(float x0, float y0, float x1, float y1, int width, int height, int iterationsCount, int *data) {
    float dX = (x1 - x0) / float(width - 1);
    float dY = (y1 - y0) / float(height - 1);
    int i;
    float x, y, Zx, Zy, tZx, tZy;
    float tmpWidth = float((blockIdx.x * blockDim.x) + threadIdx.x);
    float tmpHeight = float((blockIdx.y * blockDim.y) + threadIdx.y);

    if ((tmpWidth < (float) width) && (tmpHeight < (float) height)) {
        x = x0 + dX * tmpWidth;
        y = y0 + dY * tmpHeight;
        Zx = x;
        Zy = y;
        i = 0;

        while (i < iterationsCount && ((Zx * Zx + Zy * Zy) < 4)) {
            tZx = Zx * Zx - Zy * Zy + x;
            tZy = 2 * Zx * Zy + y;
            Zx = tZx;
            Zy = tZy;
            i++;
        }

        int index = int(tmpHeight * (float) width + tmpWidth);
        data[index] = i;
    }
}

void makePicturePNG(const int *data, int width, int height, int iterationsCount) {
    float red_value, green_value, blue_value;
    float scale = 256.0f / (float) iterationsCount;

    pngwriter png(width, height, 1.0, "mandelbrot_output.png");

    for (int j = height - 1; j >= 0; j--) {
        for (int i = 0; i < width; i++) {
            int colorIndex = (int) floor(5.0 * scale * log2f(1.0f * (float) data[j * width + i] + 1));
            red_value = colors[colorIndex][0];
            green_value = colors[colorIndex][2];
            blue_value = colors[colorIndex][1];
            png.plot(i, j, red_value, green_value, blue_value);
        }
    }

    png.close();
}

int compare(const int *data1, const int *data2, int length) {
    int sum = 0;
    int in1, in2;

    for (int i = 0; i < length; i++) {
        in1 = (data1[i] > 255) ? 1 : 0;
        in2 = (data2[i] > 255) ? 1 : 0;
        sum += (int) in1 == in2;
    }

    return sum;
}

int main(int argc, char **argv) {
    if (argc != 11) {
        printf("Wywołanie %s LD_Re, LD_Im, PG_Re, PG_Im, Poziom, Pion, Iteracje, Compare  Picture \n", argv[0]);
        printf("Flagi:  Compare: 0/1 - porównaj rezultat z CPU lub nie\n");
        printf("Flagi:  Picture: 0/1 - generuj obrazki lub nie \n");
        exit(1);
    }

    float x0 = stof(argv[1]);
    float y0 = stof(argv[2]);
    float x1 = stof(argv[3]);
    float y1 = stof(argv[4]);
    int width = stoi(argv[5]);
    int height = stoi(argv[6]);
    int iterationsCount = stoi(argv[7]);
    int shouldCompare = stoi(argv[8]);
    int shouldGenerateImage = stoi(argv[9]);
    int shouldUse2D = stoi(argv[10]);

    hipError_t status;

    int *mandel_data_host;
    int *mandel_data_device;
    int *mandel_data_cpu = (int *) malloc(sizeof(int) * width * height);

    status = hipMalloc((void **) &mandel_data_device, width * height * sizeof(int));

    if (status != hipSuccess) {
        cout << hipGetErrorString(status) << endl;
    }

    status = hipHostMalloc((void **) &mandel_data_host, width * height * sizeof(int));

    if (status != hipSuccess) {
        cout << hipGetErrorString(status) << endl;
    }

    time_t start, end;

    printf("Corners - (%lf , %lf) and ", x0, y0);
    printf("(%lf , %lf)\n", x1, y1);

    int block_width = 8;
    int block_height = 32;
    dim3 threadsPerBlock(block_width, block_height, 1);
    dim3 numBlocks(width / block_width + 1, height / block_height + 1, 1);

    start = clock();
    auto start2 = chrono::steady_clock::now();

    if (shouldUse2D) {
        cudaMandelbrot2<<<numBlocks, threadsPerBlock, 0>>>(x0, y0, x1, y1, width, height, iterationsCount,mandel_data_device);
    } else {
        cudaMandelbrot<<<numBlocks, threadsPerBlock, 0>>>(x0, y0, x1, y1, width, height, iterationsCount,mandel_data_device);
    }

    status = hipDeviceSynchronize();

    if (status != hipSuccess) {
        cout << hipGetErrorString(status) << endl;
    }

    auto stop = chrono::steady_clock::now();

    status = hipMemcpy(mandel_data_host, mandel_data_device, width * height * sizeof(int), hipMemcpyDeviceToHost);

    if (status != hipSuccess) {
        cout << hipGetErrorString(status) << endl;
    }

    end = clock();

    auto diff = stop - start2;

    cout << "Kernel " << chrono::duration<float, milli>(diff).count() << " ms" << endl;
    cout << "Kernel " << chrono::duration<float, micro>(diff).count() << " us" << endl;
    cout << "Kernel " << chrono::duration<float, nano>(diff).count() << " ns" << endl;

    printf("Start %f End %f clock ticks\n", (float) start, (float) end);
    printf("Computations and transfer %lf s\n\n", 1.0f * (float) (end - start) / CLOCKS_PER_SEC);

    if (shouldGenerateImage == 1) {
        start = clock();
        makePicturePNG(mandel_data_host, width, height, iterationsCount);
        end = clock();
        printf("Picture production took %lf s\n\n", 1.0f * (float) (end - start) / CLOCKS_PER_SEC);
    }

    status = hipFree(mandel_data_device);
    if (status != hipSuccess) {
        cout << hipGetErrorString(status) << endl;
    }

    if (shouldCompare == 1) {
        printf("Computing reference\n");
        start = clock();
//        int SUM = computeMandelbrot(x0, y0, x1, y1, width, height, iterationsCount, mandel_data_cpu);
        end = clock();
        printf("Time %lf s\n\n", 1.0f * (float) (end - start) / CLOCKS_PER_SEC);
        int ident = compare(mandel_data_host, mandel_data_cpu, height * width);
        printf("%d out of %d pixels are identical (%8.2lf) %% \n", ident, height * width, 100.0 * ident / height / width);
    }

    status = hipHostFree(mandel_data_host);

    if (status != hipSuccess) {
        cout << hipGetErrorString(status) << endl;
    }
}
